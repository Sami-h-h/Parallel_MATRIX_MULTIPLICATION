#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void initializeMatrix(int *matrix, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            matrix[i*cols+j] = rand() % 100;
        }
    }
}

void printMatrix(int* matrix, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            printf("%d ", matrix[i*cols + j]);
        }
        printf("\n");
    }
}

__global__ void matrixMultiplicationKernel (int *A, int *B, int *C, int rowsA, int colsA, int colsB){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    if((row < rowsA) && (col < colsB)){
        for(int k = 0; k < colsA; k++){
            sum += A[row*colsA+k] * B[k*colsB+col];
        }
        C[row * colsB + col] = sum;
    }
}

int main(){
    int numRowsA = 1000;
    int numColsA = 2000;
    int numColsB = 1000;

    size_t bytesA = numRowsA * numColsA * sizeof(int);
    size_t bytesC = numRowsA * numColsB * sizeof(int);

    int *A, *B, *C;

    A = (int*) malloc(bytesA);
    B = (int*) malloc(bytesA);
    C = (int*) malloc(bytesC);

    int *deviceA, *deviceB, *deviceC;

    hipMalloc(&deviceA, bytesA);
    hipMalloc(&deviceB, bytesA);
    hipMalloc(&deviceC, bytesC);

    initializeMatrix(A, numRowsA, numColsA);
    initializeMatrix(B, numColsA, numColsB);

    hipMemcpy(deviceA, A, bytesA, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B, bytesA, hipMemcpyHostToDevice);

    int blockSize = 16;
    int numBlocks = (int)ceil((float)numRowsA / blockSize);

    dim3 grid(numBlocks, numBlocks);
    dim3 threads(blockSize, blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, NULL);

    matrixMultiplicationKernel <<<grid, threads>>> (deviceA, deviceB, deviceC, numRowsA, numColsA, numColsB);

    hipMemcpy(C, deviceC, bytesC, hipMemcpyDeviceToHost);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time in GPU: %.2f ms\n", milliseconds);
    printf("Number of threads: %d\n", numBlocks * numBlocks * blockSize * blockSize);

    printMatrix(C, numRowsA, numColsB);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    free(A);
    free(B);
    free(C);

    return 0;
}
